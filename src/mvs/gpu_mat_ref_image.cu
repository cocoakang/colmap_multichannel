#include "hip/hip_runtime.h"
// Copyright (c) 2018, ETH Zurich and UNC Chapel Hill.
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.
//
//     * Redistributions in binary form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in the
//       documentation and/or other materials provided with the distribution.
//
//     * Neither the name of ETH Zurich and UNC Chapel Hill nor the names of
//       its contributors may be used to endorse or promote products derived
//       from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDERS OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.
//
// Author: Johannes L. Schoenberger (jsch-at-demuc-dot-de)

#include "mvs/gpu_mat_ref_image.h"

#include <iostream>

#include "util/cudacc.h"
#define TMP_DEPTH 3
#define MAX_DEPTH 70
namespace colmap {
	namespace mvs {
		namespace {

			texture<uint8_t, hipTextureType2DLayered, hipReadModeNormalizedFloat> image_texture;

			__global__ void FilterKernel(GpuMat<uint8_t> image, GpuMat<float> sum_image,
				GpuMat<float> squared_sum_image,
				const int window_radius, const int window_step,
				const float sigma_spatial,
				const float sigma_color,
				const int data_depth) {
				const size_t row = blockDim.y * blockIdx.y + threadIdx.y;
				const size_t col = blockDim.x * blockIdx.x + threadIdx.x;
				if (row >= image.GetHeight() || col >= image.GetWidth()) {
					return;
				}

				BilateralWeightComputer bilateral_weight_computer(sigma_spatial, sigma_color);

				//const float center_color = tex2D(image_texture, col, row);
				//TODO-x: now is grey
				//float center_color_aver = 0;
				//for (int i = 0; i < data_depth; ++i)
				//	center_color_aver += tex2DLayered(image_texture, col, row, i);
				//center_color_aver /= data_depth;
				float center_color[MAX_DEPTH];
				for (int i = 0; i < data_depth; ++i)
					center_color[i] = tex2DLayered(image_texture, col, row, i);


				//float color_sum = 0.0f;
				//float color_squared_sum = 0.0f;
				//NEW
				float bilateral_weight_sum = 0.0f;

				float color_sum[MAX_DEPTH];
				float color_squared_sum[MAX_DEPTH];
				for (int i = 0; i < TMP_DEPTH; ++i)
				{
					color_sum[i] = 0;
					color_squared_sum[i] = 0;
				}

				for (int window_row = -window_radius; window_row <= window_radius;
					window_row += window_step) {
					for (int window_col = -window_radius; window_col <= window_radius;
						window_col += window_step) {
						//const float color =
						//	tex2D(image_texture, col + window_col, row + window_row);
						//float color_aver = 0;
						//for (int i = 0; i < data_depth; ++i)
						//	color_aver += tex2DLayered(image_texture, col + window_col, row + window_row, i);
						//color_aver /= data_depth;

						//NEW
						float color[MAX_DEPTH];
						for (int i = 0; i < TMP_DEPTH; ++i)
							color[i] = tex2DLayered(image_texture, col + window_col, row + window_row, i);

						const float bilateral_weight = bilateral_weight_computer.Compute(
							window_row, window_col, center_color, color, TMP_DEPTH); //NEW

						//color_sum += bilateral_weight * color;
						//color_squared_sum += bilateral_weight * color * color;
						bilateral_weight_sum += bilateral_weight;
						//NEW
						for (int i = 0; i < TMP_DEPTH; ++i)
						{
							color_sum[i] += bilateral_weight * color[i];
							color_squared_sum[i] += bilateral_weight * color[i] * color[i];
						}
					}
				}

				//color_sum /= bilateral_weight_sum;
				//color_squared_sum /= bilateral_weight_sum;
				//NEW
				for (int i = 0; i < TMP_DEPTH; ++i)
				{
					color_sum[i] /= bilateral_weight_sum;
					color_squared_sum[i] /= bilateral_weight_sum;
				}


				//image.Set(row, col, static_cast<uint8_t>(255.0f * center_color));//CHANGED
				//sum_image.Set(row, col, color_sum);
				//squared_sum_image.Set(row, col, color_squared_sum);
				//NEW
				for (int i = 0; i < data_depth; ++i)
				{
					image.Set(row, col, i, static_cast<uint8_t>(255.0f * tex2DLayered(image_texture, col, row, i)));
					sum_image.Set(row, col, i, color_sum[i]);
					squared_sum_image.Set(row, col, i, color_squared_sum[i]);
				}
			}

		}  // namespace

		GpuMatRefImage::GpuMatRefImage(const size_t width, const size_t height, const size_t depth)
			: height_(height), width_(width), depth_(depth) {
			image.reset(new GpuMat<uint8_t>(width, height, depth));
			sum_image.reset(new GpuMat<float>(width, height, depth)); //NEW
			squared_sum_image.reset(new GpuMat<float>(width, height, depth));//NEW
		}

		void GpuMatRefImage::Filter(const uint8_t* image_data,
			const size_t window_radius,
			const size_t window_step, const float sigma_spatial,
			const float sigma_color) {
			CudaArrayWrapper<uint8_t> image_array(width_, height_, depth_);
			image_array.CopyToDevice(image_data);
			image_texture.addressMode[0] = hipAddressModeBorder;  //out of address return 0
			image_texture.addressMode[1] = hipAddressModeBorder;
			image_texture.addressMode[2] = hipAddressModeBorder;
			// returned value is the texel whose texture coordinates are the closest to
			// the input texture coordinates
			image_texture.filterMode = hipFilterModePoint;
			image_texture.normalized = false;

			const dim3 block_size(kBlockDimX, kBlockDimY);
			const dim3 grid_size((width_ - 1) / block_size.x + 1,
				(height_ - 1) / block_size.y + 1);

			CUDA_SAFE_CALL(hipBindTextureToArray(image_texture, image_array.GetPtr()));
			FilterKernel << <grid_size, block_size >> >(
				*image, *sum_image, *squared_sum_image, window_radius, window_step,
				sigma_spatial, sigma_color, depth_);
			CUDA_SYNC_AND_CHECK();
			CUDA_SAFE_CALL(hipUnbindTexture(image_texture));
		}

	}  // namespace mvs
}  // namespace colmap
